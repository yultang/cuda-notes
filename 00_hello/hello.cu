/* hello.cu */

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world(void) {
    printf("GPU: Hello, world!\n");
}

int main(int argc, char *argv[]) {
    printf("CPU: Hello, world!\n");
    hello_world<<<1, 10>>>();
    /*
    cudaError_t error_code = cudaGetLastError();
    if (error_code != cudaSuccess) {
        printf("CUDA Error: %s", cudaGetErrorString(error_code));
        exit(EXIT_FAILURE);
    }
    cudaDeviceSynchronize();
    */
    hipDeviceReset();
    return 0;
}