/* check_dimension.cu */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void checkIndex(void) {
    printf("threadIdx: (%d, %d, %d)\n"
           "blockIdx:  (%d, %d, %d)\n"
           "blockDim:  (%d, %d, %d)\n"
           "gridDim:   (%d, %d, %d)\n\n",
            threadIdx.x, threadIdx.y, threadIdx.z,
            blockIdx.x, blockIdx.y, blockIdx.z,
            blockDim.x, blockDim.y, blockDim.z,
            gridDim.x, gridDim.y, gridDim.z);
}

int main(int argc, char *argv[]) {
    // 假定的要处理的元素总数
    int nElem = 6;
    dim3 block(3);
    // (nElem + block.x - 1) / block.x 是一种数学技巧
    // 实际原理是通过向上取整来获得要使用的grid数量:
    // 1. 当nElem能被block.x整除时, 
    //      (nElem + block.x - 1) / block.x = nElem / block.x 
    // 2. 不能整除时, 
    //      (nElem + block.x - 1) / block.x = nElem / block.x + 1

    dim3 grid((nElem + block.x - 1) / block.x);

    printf("grid.x: %d, grid.y: %d, grid.z: %d\n", grid.x, grid.y, grid.z);
    printf("block.x: %d, block.y: %d, block.z: %d\n", block.x, block.y, block.z);
    checkIndex<<<grid, block>>>();
    hipDeviceReset();
    return 0;
}